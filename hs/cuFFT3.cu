#include <stdio.h>
#include <hipfft/hipfft.h>
#include<hip/hip_runtime.h>
#define LEN 2048
#define SQRT2048_real 45.2876
#define SQRT2048_imag 45.3065

__global__ void int_cufftComplex(int16_t *a, hipfftComplex *b, int length)
{
    int id = (blockIdx.x * blockDim.x + threadIdx.x);
    if(id >=length)
    {
        return;
    }
    b[id].x = a[id*2];
    b[id].y = a[id*2+1];
}

__global__ void cufftComplex_int(hipfftComplex *a, int16_t *b, int length)
{
    int id = (blockIdx.x * blockDim.x + threadIdx.x);
    if(id >=length)
    {
        return;
    }
    b[id*2] = a[id].x/SQRT2048_real;
    b[id*2+1] = a[id].y/SQRT2048_imag;
    // b[id*2] = a[id].x;
    // b[id*2+1] = a[id].y;
}


int16_t *x1;
hipfftComplex *CompData;
hipfftHandle plan;
void initcudft()
{
    hipMalloc((void**)&x1, LEN * sizeof(int32_t));
    hipMalloc((void**)&CompData, LEN * sizeof(hipfftComplex));
    hipfftPlan1d(&plan, LEN, HIPFFT_C2C, 1);//declaration,这句要warm-up
}
void destroycudft()
{
    // cudaFree(CompData);
    // cudaFree(x1);
    // cufftDestroy(plan);
}

void cudft2048(int16_t *x,int16_t *y,unsigned char scale)
{

    // int16_t *x1;
    // cudaMalloc((void**)&x1, LEN * sizeof(int32_t));
    hipMemcpy(x1, x, LEN * sizeof(int32_t), hipMemcpyHostToDevice);


    int threadNum = 512;
    int blockNum = 4;
    // cufftComplex *CompData;
    // cudaMalloc((void**)&CompData, LEN * sizeof(cufftComplex));

    int_cufftComplex<<<blockNum, threadNum>>>(x1, CompData, LEN);
    // int_cufftComplex<<<1, 8>>>(x1, CompData, LEN);

    // cufftHandle plan;// cuda library function handle
    // cufftPlan1d(&plan, LEN, CUFFT_C2C, 1);//declaration,这句要warm-up
    hipfftExecC2C(plan, (hipfftComplex*)CompData, (hipfftComplex*)CompData, HIPFFT_FORWARD);//execute
    hipDeviceSynchronize();//wait to be done

    cufftComplex_int<<<blockNum, threadNum>>>(CompData, x1, LEN);
    // cufftComplex_int<<<1, 8>>>(CompData, x1, LEN);
    hipMemcpy(y, x1, LEN * sizeof(int32_t), hipMemcpyDeviceToHost);// copy the result from device to host

    // printf("hs1111111111111111:\n");

    // cufftDestroy(plan);
    // cudaFree(CompData);
    // cudaFree(x1);
}

void load_cuFFT(void) 
{
    initcudft();
    int16_t *a = (int16_t *)malloc(LEN * sizeof(int32_t));
    int16_t *b = (int16_t *)malloc(LEN * sizeof(int32_t));
    cudft2048(a,b,1);
}

int main()
{
    load_cuFFT();
    int16_t *a = (int16_t *)malloc(LEN * sizeof(int32_t));
    int16_t *b = (int16_t *)malloc(LEN * sizeof(int32_t));
    for (int i = 0; i < LEN; i++)
    {
        *(a+2*i) = i;
        *(a+2*i+1) = LEN-i;
    }
    for (int i = 0; i < 3; i++)
    {
        cudft2048((int16_t *)a,(int16_t *)b,0);
        printf("hs1111111111111111:\n");
        for (int j = 0; j < LEN; j++)
        {
            printf("a=%d + %dj\tb=%d + %dj\n", a[j*2],a[j*2+1],b[j*2],b[j*2+1]);
        }
    }
    destroycudft();

}