#include <stdio.h>
#include <stdlib.h>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>

#define SIZE  1024*1024

char *gpu;
// int16_t *cuda_temp;
void init_cuda()
{
    hipMalloc((void**)&gpu, SIZE * sizeof(char));
}

FILE *fp;
FILE *fp1;
FILE *fp2;
void cudamemcpy(char *x,char *y,int z)
{
    hipEvent_t start, stop;
    float time;
    hipMemcpy(gpu, x, z * sizeof(char), hipMemcpyHostToDevice);
    hipDeviceSynchronize();//wait to be done
    hipEventRecord( start, 0 );
    hipMemcpy(y, gpu, z * sizeof(char), hipMemcpyDeviceToHost);
    hipEventRecord( stop, 0 );
    hipEventSynchronize(start);
    hipEventSynchronize( stop );//注意函数所处位置
    hipEventElapsedTime( &time, start, stop );
    fprintf(fp2,"%.8f\n",time*1000);
}


int main()
{
    if((fp=fopen("time.txt","w"))==NULL)
        printf("Cannot open .\n");
    if((fp1=fopen("time1.txt","w"))==NULL)
        printf("Cannot open .\n");
    if((fp2=fopen("time2.txt","w"))==NULL)
        printf("Cannot open .\n");
    hipEventCreate(&start);
    hipEventCreate(&stop);
    init_cuda();
    char *cpu1,*cpu2;
    int a;
    hipHostAlloc((void **)&cpu1, SIZE * sizeof(char), hipHostMallocDefault);
    hipHostAlloc((void **)&cpu2, SIZE * sizeof(char), hipHostMallocDefault);
    for (int i = 0; i < SIZE; i++)
    {
        *cpu1 =rand();
        cpu1++;
    }
    
    a=1;
    for (int i = 1; i < 1024; i++)
    {
        a =i;
        printf("%d\t\t",a);
        hipEventRecord( start, 0 );
        cudamemcpy(cpu1,cpu2,a);
        hipEventRecord( stop, 0 );
        hipEventSynchronize(start);
        hipEventSynchronize( stop );//注意函数所处位置
        hipEventElapsedTime( &time, start, stop );
        printf("cudft2048执行时间：%f(us)\n",time*1000);
        fprintf(fp,"%.8f\n",time*1000);
    }
    for (int i = 1; i < 1024; i++)
    {
        a = i;
        fprintf(fp1,"%d\n",a);
    }*/
    for (int i = 1; i < 1024; i++)
    {
        a =i*1024;
        cudamemcpy(cpu1,cpu2,a);
    }
}