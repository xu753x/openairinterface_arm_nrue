#include <stdio.h>
#include <stdlib.h>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>

#define SIZE  1024*1024

char *gpu;
// int16_t *cuda_temp;
void init_cuda()
{
    hipMalloc((void**)&gpu, SIZE * sizeof(char));
}

FILE *fp;
FILE *fp1;
FILE *fp2;
void cudamemcpy(char *x,char *y,int z)
{
    hipEvent_t start, stop;
    float time;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipMemcpy(gpu, x, z * sizeof(char), hipMemcpyHostToDevice);
    hipDeviceSynchronize();//wait to be done
    hipEventRecord( start, 0 );
    hipMemcpy(y, gpu, z * sizeof(char), hipMemcpyDeviceToHost);
    hipEventRecord( stop, 0 );
    hipEventSynchronize(start);
    hipEventSynchronize( stop );//注意函数所处位置
    hipEventElapsedTime( &time, start, stop );
    fprintf(fp2,"%.8f\n",time*1000);
}


int main()
{
    if((fp=fopen("time.txt","w"))==NULL)
        printf("Cannot open .\n");
    if((fp1=fopen("time1.txt","w"))==NULL)
        printf("Cannot open .\n");
    if((fp2=fopen("time2.txt","w"))==NULL)
        printf("Cannot open .\n");
    init_cuda();
    char *cpu1,*cpu2;
    int a;
    hipHostAlloc((void **)&cpu1, SIZE * sizeof(char), hipHostMallocDefault);
    hipHostAlloc((void **)&cpu2, SIZE * sizeof(char), hipHostMallocDefault);
    for (int i = 0; i < SIZE; i++)
    {
        *cpu1 =rand();
        cpu1++;
    }
    
    for (int i = 1; i < 1024; i++)
    {
        a =i*1024;
        cudamemcpy(cpu1,cpu2,a);
    }
}