#include <stdio.h>
#include <hipfft/hipfft.h>
#include<hip/hip_runtime.h>
#define LEN 2048

__global__ void int_cufftComplex(int16_t *a, hipfftComplex *b, int length)
{
    int id = (blockIdx.x * blockDim.x + threadIdx.x);
    if(id >=length)
    {
        return;
    }
    b[id].x = a[id*2];
    b[id].y = a[id*2+1];
}

__global__ void cufftComplex_int(hipfftComplex *a, int16_t *b, int length)
{
    int id = (blockIdx.x * blockDim.x + threadIdx.x);
    if(id >=length)
    {
        return;
    }
    b[id*2] = a[id].x/45.2;
    b[id*2+1] = a[id].y/45.2;
}

void initcudft()
{
}

void cudft2048(int16_t *x,int16_t *y,unsigned char scale)
{

    int16_t *x1;
    hipMalloc((void**)&x1, LEN * sizeof(int32_t));
    hipMemcpy(x1, x, LEN * sizeof(int32_t), hipMemcpyHostToDevice);

    int threadNum = 512;
    int blockNum = 4;
    hipfftComplex *CompData;
    hipMalloc((void**)&CompData, LEN * sizeof(hipfftComplex));
    int_cufftComplex<<<blockNum, threadNum>>>(x1, CompData, LEN);

    hipfftHandle plan;// cuda library function handle
    hipfftPlan1d(&plan, LEN, HIPFFT_C2C, 1);//declaration,这句要warm-up
    hipfftExecC2C(plan, (hipfftComplex*)CompData, (hipfftComplex*)CompData, HIPFFT_FORWARD);//execute
    hipDeviceSynchronize();//wait to be done

    
    cufftComplex_int<<<blockNum, threadNum>>>(CompData, x1, LEN);
    hipMemcpy(y, x1, LEN * sizeof(int32_t), hipMemcpyDeviceToHost);// copy the result from device to host

    // printf("hs1111111111111111:\n");
    // for (int i = 0; i < LEN; i++)
    // {
    //     printf("a=%d + %dj\tb=%d + %dj\n", x[i*2],x[i*2+1],y[i*2],y[i*2+1]);
    // }

    hipfftDestroy(plan);
    hipFree(CompData);
    hipFree(x1);
}

int load_cuFFT(void) 
{
    initcudft();
    cudft2048(0,0,1);
    return 0;
}

int main()
{
    load_cuFFT();
    int16_t *a = (int16_t *)malloc(LEN * sizeof(int32_t));
    int16_t *b = (int16_t *)malloc(LEN * sizeof(int32_t));
    int i;
    for (i = 0; i < LEN; i++)
    {
        *(a+2*i) = i;
        *(a+2*i+1) = LEN-i;
    }
    for (i = 0; i < 3; i++)
    {
        cudft2048((int16_t *)a,(int16_t *)b,0);
    }

        

}