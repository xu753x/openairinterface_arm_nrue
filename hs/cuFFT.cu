#include <stdio.h>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>

#define LEN 2048
#define SQRT2048_real 45.2876
#define SQRT2048_imag 45.3065
#define SYMBOLS_PER_SLOT 14

__global__ void int_cufftComplex(int16_t *a, hipfftComplex *b, int length)
{
    int id = (blockIdx.x * blockDim.x + threadIdx.x);
    if(id >=length)
    {
        return;
    }
    b[id].x = a[id*2];
    b[id].y = a[id*2+1];
}

__global__ void cufftComplex_int(hipfftComplex *a, int16_t *b, int length)
{
    int id = (blockIdx.x * blockDim.x + threadIdx.x);
    if(id >=length)
    {
        return;
    }
    b[id*2] = a[id].x/SQRT2048_real;
    b[id*2+1] = a[id].y/SQRT2048_imag;
}


int16_t *x1;
hipfftComplex *CompData;
hipfftHandle plan;
void initcudft()
{
    hipMalloc((void**)&x1, SYMBOLS_PER_SLOT*LEN * sizeof(int32_t));
    hipMalloc((void**)&CompData, SYMBOLS_PER_SLOT*LEN * sizeof(hipfftComplex));
    // cufftPlan1d(&plan, LEN, CUFFT_C2C, 1);
	int rank=1;
	int n[1]; n[0]=LEN;
    int nembed[2]; nembed[0]=LEN; nembed[1]=SYMBOLS_PER_SLOT;
    int stride=1;
    int dist = LEN;
    int batch=SYMBOLS_PER_SLOT;
    hipfftPlanMany(&plan,rank,n,nembed, stride ,dist , nembed, stride,dist, HIPFFT_C2C, batch);
}

void cudft2048(int16_t *x,int16_t *y,unsigned char scale)
{

    hipMemcpy(x1, x, SYMBOLS_PER_SLOT*LEN * sizeof(int32_t), hipMemcpyHostToDevice);

    int threadNum = 512;
    int blockNum = (SYMBOLS_PER_SLOT * LEN - 0.5) / threadNum + 1;
    int_cufftComplex<<<blockNum, threadNum>>>(x1, CompData, SYMBOLS_PER_SLOT*LEN);

    hipfftExecC2C(plan, (hipfftComplex*)CompData, (hipfftComplex*)CompData, HIPFFT_FORWARD);//execute
    hipDeviceSynchronize();//wait to be done

    cufftComplex_int<<<blockNum, threadNum>>>(CompData, x1, SYMBOLS_PER_SLOT*LEN);
    hipMemcpy(y, x1, SYMBOLS_PER_SLOT*LEN * sizeof(int32_t), hipMemcpyDeviceToHost);// copy the result from device to host
    static int hshs=0;
    printf("------------%d\n",hshs);
    hshs++;
}

void load_cuFFT(void) 
{
    initcudft();
    int16_t *a = (int16_t *)malloc(SYMBOLS_PER_SLOT*LEN * sizeof(int32_t));
    int16_t *b = (int16_t *)malloc(SYMBOLS_PER_SLOT*LEN * sizeof(int32_t));
    cudft2048(a,b,1);
}



int main()
{
    load_cuFFT();
    int16_t *a = (int16_t *)malloc(SYMBOLS_PER_SLOT*LEN * sizeof(int32_t));
    int16_t *b = (int16_t *)malloc(SYMBOLS_PER_SLOT*LEN * sizeof(int32_t));
    for (int i = 0; i < SYMBOLS_PER_SLOT*LEN; i++)
    {
        *(a+2*i) = i;
        *(a+2*i+1) = LEN-i;
    }
    for (int i = 0; i < 3; i++)
    {
        hipEvent_t start, stop;
        float time;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord( start, 0 );
        cudft2048((int16_t *)a,(int16_t *)b,0);
        hipEventRecord( stop, 0 );
        hipEventSynchronize(start);
        hipEventSynchronize( stop );//注意函数所处位置
        hipEventElapsedTime( &time, start, stop );
        printf("cudft2048执行时间：%f(us)\n",time*1000);
        // printf("hs1111111111111111:\n");
        // for (int j = 0; j < SYMBOLS_PER_SLOT*LEN; j++)
        // {
        //     printf("a=%d + %dj\tb=%d + %dj\n", a[j*2],a[j*2+1],b[j*2],b[j*2+1]);
        // }
    }
}