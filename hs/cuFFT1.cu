
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hipfft/hipfft.h>

//gdb debug
// void testtest(int16_t *x,int16_t *y,unsigned char scale)
// {
//     printf("testtest \n");
// }
#define LEN 2048 //signal sampling points
void cudft2048(int16_t *x,int16_t *y,unsigned char scale)
{
    // testtest(0,0,0);
    // printf("hs222222：\n");
    hipfftComplex *CompData = (hipfftComplex*)malloc(LEN * sizeof(hipfftComplex));//allocate memory for the data in host
    hipfftComplex *CompData1 = (hipfftComplex*)malloc(LEN * sizeof(hipfftComplex));
    for (int i = 0; i < LEN; i++)
    {
        // printf("%d\n",i);
        CompData[i].x = x[i*2];
        CompData[i].y = x[i*2+1];
    }

    hipfftComplex *d_fftData;
    hipMalloc((void**)&d_fftData, LEN * sizeof(hipfftComplex));// allocate memory for the data in device
    hipMemcpy(d_fftData, CompData, LEN * sizeof(hipfftComplex), hipMemcpyHostToDevice);// copy data from host to device
    
    hipfftHandle plan;// cuda library function handle
    hipfftPlan1d(&plan, LEN, HIPFFT_C2C, 1);//declaration
    hipfftExecC2C(plan, (hipfftComplex*)d_fftData, (hipfftComplex*)d_fftData, HIPFFT_FORWARD);//execute
    hipDeviceSynchronize();//wait to be done
    hipMemcpy(CompData1, d_fftData, LEN * sizeof(hipfftComplex), hipMemcpyDeviceToHost);// copy the result from device to host

    for (int i = 0; i < LEN; i++)
    {
        y[i*2] = CompData1[i].x/45.2;
        y[i*2+1] = CompData1[i].y/45.2;
    }
    // printf("hs1111111111111111:\n");
    // for (int i = 0; i < LEN; i++)
    // {
    //     printf("a=%d + %dj\tb=%d + %dj\n", x[i*2],x[i*2+1],y[i*2],y[i*2+1]);
    // }
    hipfftDestroy(plan);
    free(CompData);
    hipFree(d_fftData);

}
void initcudft()
{
}
int main()
{
    int16_t *a = (int16_t *)malloc(LEN * sizeof(int32_t));
    int i;
    for (i = 0; i < LEN; i++)
    {
        *(a+2*i) = i;
        *(a+2*i+1) = LEN-i;
    }
    for (i = 0; i < 3; i++)
    {
        int32_t *b = (int32_t *)malloc(LEN * sizeof(int32_t));
        cudft2048((int16_t *)a,(int16_t *)b,1);
        free(b);
    }
    
}