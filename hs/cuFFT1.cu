#include <stdio.h>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>

#define LEN 2048
#define SQRT2048_real 45.2876
#define SQRT2048_imag 45.3065
#define SYMBOLS_PER_SLOT 14

__global__ void int_cufftComplex(int16_t *a, hipfftComplex *b, int length)
{
    int id = (blockIdx.x * blockDim.x + threadIdx.x);
    if(id >=length)
    {
        return;
    }
    b[id].x = a[id*2];
    b[id].y = a[id*2+1];
}

__global__ void cufftComplex_int(hipfftComplex *a, int16_t *b, int length)
{
    int id = (blockIdx.x * blockDim.x + threadIdx.x);
    if(id >=length)
    {
        return;
    }
    b[id*2] = a[id].x/SQRT2048_real;
    b[id*2+1] = a[id].y/SQRT2048_imag;
}

int16_t *cuda_x;
int16_t *cuda_y;
int16_t *x11;
hipfftComplex *CompData1;
hipfftHandle plan1;
void initcudft()
{
    hipMalloc((void**)&x11, SYMBOLS_PER_SLOT*LEN * sizeof(int32_t));
    hipMalloc((void**)&CompData1, SYMBOLS_PER_SLOT*LEN * sizeof(hipfftComplex));
    // cufftPlan1d(&plan1, LEN, CUFFT_C2C, SYMBOLS_PER_SLOT);
	int rank=1;
	int n[1]; n[0]=LEN;
    int nembed[2]; nembed[0]=LEN; nembed[1]=SYMBOLS_PER_SLOT;
    int stride=1;
    int dist = LEN;
    int batch=SYMBOLS_PER_SLOT;
    hipfftPlanMany(&plan1,rank,n,nembed, stride ,dist , nembed, stride,dist, HIPFFT_C2C, batch);
    hipHostAlloc((void **)&cuda_x, SYMBOLS_PER_SLOT*LEN * sizeof(int32_t), hipHostMallocDefault);
    hipHostAlloc((void **)&cuda_y, SYMBOLS_PER_SLOT*LEN * sizeof(int32_t), hipHostMallocDefault);
}

void cudft2048(int16_t *x,int16_t *y,unsigned char scale)
{
    // cudaStream_t stream;
	// cudaStreamCreate(&stream);

    hipEvent_t start, stop;
    float time;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord( start, 0 );

    memcpy(cuda_x,x,SYMBOLS_PER_SLOT*LEN * sizeof(int32_t));
    hipMemcpy(x11, cuda_x, SYMBOLS_PER_SLOT*LEN * sizeof(int32_t), hipMemcpyHostToDevice);

    int threadNum = 512;
    int blockNum = (SYMBOLS_PER_SLOT * LEN - 1) / threadNum + 1;
    
    int_cufftComplex<<<blockNum, threadNum>>>(x11, CompData1, SYMBOLS_PER_SLOT*LEN);

    hipfftExecC2C(plan1, (hipfftComplex*)CompData1, (hipfftComplex*)CompData1, HIPFFT_FORWARD);//execute
    hipDeviceSynchronize();//wait to be done

    cufftComplex_int<<<blockNum, threadNum>>>(CompData1, x11, SYMBOLS_PER_SLOT*LEN);
    hipMemcpy(cuda_y, x11, SYMBOLS_PER_SLOT*LEN * sizeof(int32_t), hipMemcpyDeviceToHost);// copy the result from device to host
    memcpy(y,cuda_y,SYMBOLS_PER_SLOT*LEN * sizeof(int32_t));

    hipEventRecord( stop, 0 );
    hipEventSynchronize(start);
    hipEventSynchronize( stop );//注意函数所处位置
    hipEventElapsedTime( &time, start, stop );
    printf("cudft2048执行时间：%f(us)\n",time*1000);

}

void load_cuFFT(void) 
{
    initcudft();
    int16_t *a = (int16_t *)malloc(SYMBOLS_PER_SLOT*LEN * sizeof(int32_t));
    int16_t *b = (int16_t *)malloc(SYMBOLS_PER_SLOT*LEN * sizeof(int32_t));
    cudft2048(a,b,1);
}



int main()
{
    load_cuFFT();
    int16_t *a = (int16_t *)malloc(SYMBOLS_PER_SLOT*LEN * sizeof(int32_t));
    int16_t *b = (int16_t *)malloc(SYMBOLS_PER_SLOT*LEN * sizeof(int32_t));
    // int16_t *a;
    // int16_t *b;
    // cudaHostAlloc((void **)&a, SYMBOLS_PER_SLOT*LEN * sizeof(int32_t), cudaHostAllocDefault);
    // cudaHostAlloc((void **)&b, SYMBOLS_PER_SLOT*LEN * sizeof(int32_t), cudaHostAllocDefault);
    for (int i = 0; i < SYMBOLS_PER_SLOT*LEN; i++)
    {
        *(a+2*i) = rand()%LEN;
        *(a+2*i+1) = rand()%LEN;
    }
    for (int i = 0; i < 10; i++)
    {
        cudft2048((int16_t *)a,(int16_t *)b,0);
        // printf("hs1111111111111111:\n");
        // for (int j = 0; j < SYMBOLS_PER_SLOT*LEN; j++)
        // {
        //     printf("a=%d + %dj\tb=%d + %dj\n", a[j*2],a[j*2+1],b[j*2],b[j*2+1]);
        // }
    }
}
